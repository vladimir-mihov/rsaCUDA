#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <math.h>
#include <vector>
#include "mandelbrotData.hpp"
#include "parser/parser.hpp"
#include "lodepng/lodepng.h"

#define NOW chrono::high_resolution_clock::now()

using uchar = unsigned char;
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUAssert: %s %s %d\n", hipGetErrorString(code), file, line);
   }
}

__global__ void mandelbrot( mandelbrotData *data, uchar *result ) {
	int	index = blockDim.x*blockIdx.x + threadIdx.x,
		col = index % data->width,
		row = index / data->width;


	if( col > data->width-1 || row > data->height-1 ) return;

	double	c_re = data->startX + col*data->stepX,
			c_im = data->startY - row*data->stepY,
			x = 0, y = 0;

	uchar iterations = 0;

	while( x*x+y*y < 4 && iterations < 255 )
	{
		double expX = exp(x), sinY, cosY;
		sincos(y,&sinY,&cosY);
		
		double xNew = expX*cosY - c_re;
		y = expX*sinY - c_im;
		x = xNew;
		iterations++;
	}
	result[index] = iterations;
}

#ifdef DRAW
void writePNG( uchar *result, string& outputFilename, mandelbrotData& data );
#endif

int main( int argc, char ** argv ) {
	// initial variables
	uchar *result, *d_result;
	mandelbrotData data, *d_data;
	int threadsPerBlock;
	string outputFilename;
	bool verbose;

	// initialize variables without leaving junk
	{
		parser cmd;
		programOptions opts;
		try {
			opts = cmd.parse(argc,argv);
		} catch ( const exception& e ) {
			cerr << e.what();
			return 1;
		}
		data = mandelbrotData( opts.width, opts.height, opts.startX, opts.endX, opts.startY, opts.endY );
		threadsPerBlock = opts.tCount;
		outputFilename = opts.outputFilename;
		verbose = opts.verbose;
	}
	cout << (verbose ? "Done parsing command line arguments.\n" : "");

	result = new uchar[data.pixels];

	cout << (verbose ? "Allocating memory on the GPU.\n" : "");
	gpuErrchk( hipMalloc((void **)&d_result, data.pixels*sizeof(uchar)) );
	gpuErrchk( hipMalloc((void **)&d_data, sizeof(mandelbrotData)) );

	gpuErrchk( hipMemcpy( d_data, &data, sizeof(mandelbrotData), hipMemcpyHostToDevice ) );

	cout << (verbose ? "Calculating mandelbrot.\n" : "");
	auto t1 = NOW;
	mandelbrot<<<(data.pixels+threadsPerBlock-1)/threadsPerBlock,threadsPerBlock>>>(d_data,d_result);
	hipDeviceSynchronize();
	auto t2 = NOW;
	cout << (verbose ? "Done. It took " : "") << chrono::duration<double,milli>(t2-t1).count() << " ms.\n";

	gpuErrchk( hipMemcpy( result, d_result, data.pixels*sizeof(uchar), hipMemcpyDeviceToHost ) );

#ifdef DRAW
	cout << (verbose ? "Generating png image.\n" : "");
	auto t3 = NOW;
	writePNG( result, outputFilename, data );
	auto t4 = NOW;
	if( verbose )
		cout << "Done. It took " << chrono::duration<double,milli>(t4-t3).count() << " ms.\n";
#endif

	gpuErrchk( hipFree(d_result) );
	gpuErrchk( hipFree(d_data) );
	delete[] result;

	return 0;
}

#ifdef DRAW
void writePNG( uchar *result, string& outputFilename, mandelbrotData& data )
{
	int w = data.width, h = data.height;
	vector<uchar> rawPixelData(w*h*4);
	for (int y = 0; y < h; ++y)
		for (int x = 0; x < w; ++x)
		{
			int index = 4*w*y + 4*x;
			uchar resultElement = result[y*w+x];
			rawPixelData[index] = resultElement == 255 ? 0 : resultElement+17;
			rawPixelData[index+1] = resultElement == 255 ? 0 : resultElement+20;
			rawPixelData[index+2] = resultElement == 255 ? 0 : resultElement+40;
			rawPixelData[index+3] = 255;
		}
	unsigned int error = lodepng::encode( outputFilename.c_str(), rawPixelData, w, h );
	if( error ) cerr << "encoder error " << error << ": " << lodepng_error_text(error) << endl;
}
#endif
